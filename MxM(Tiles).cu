
#include <hip/hip_runtime.h>
#include <stdio.h>

/*	Multiplicación de Matrices
 *
 * Integrantes:
 *	-Fabricio Ballón Cuadros.
 *	-Angel Suacapuca Díaz.
*/

#define TILE_WIDTH 2

__global__ void MatrixMulKernel1(float* d_M, float* d_N, float* d_P, int Width) {
	// Calculate the row index of the d_P element and d_M
	int Row = blockIdx.y*blockDim.y+threadIdx.y;
	// Calculate the column index of d_P and d_N
	int Col = blockIdx.x*blockDim.x+threadIdx.x;
	if ((Row < Width) && (Col < Width)) {
		float Pvalue = 0;
		// each thread computes one element of the block sub-matrix
		for (int k = 0; k < Width; ++k) {
			Pvalue += d_M[Row*Width+k]*d_N[k*Width+Col];
		}
		d_P[Row*Width+Col] = Pvalue;
	}
}

__global__ void MatrixMulKernel(float *d_M, float *d_N, float *d_P, int Width) {
	__shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
	__shared__ float Nds[TILE_WIDTH][TILE_WIDTH];
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	// Identify the row and column of the d_P element to work on
	int Row = by * TILE_WIDTH + ty;
	int Col = bx * TILE_WIDTH + tx;
	float Pvalue = 0;

	// Loop over the d_M and d_N tiles required to compute d_P element
	for (int m = 0; m < Width / TILE_WIDTH; ++m) {
	// Coolaborative loading of d_M and d_N tiles into shared memory
		Mds[ty][tx] = d_M[Row * Width + m * TILE_WIDTH + tx];
		Nds[ty][tx] = d_N[(m * TILE_WIDTH + ty) * Width + Col];
		__syncthreads();
		for (int k = 0; k < TILE_WIDTH; ++k) {
			Pvalue += Mds[ty][k] * Nds[k][tx];
		}
		__syncthreads();
	}
	d_P[Row * Width + Col] = Pvalue;
}

__host__ int main(int argc, char const *argv[]) {

	float *h_M;
	float *h_N;
	float *h_P;
	float *d_M;
	float *d_N;
	float *d_P;
	int width = atoi(argv[1]);
	int size = width * width * sizeof(float);

	h_M = (float*)malloc(size);
	h_N = (float*)malloc(size);
	h_P = (float*)malloc(size);

	for (size_t i = 0; i < width*width; i++) {
		h_M[i] = 1;
		h_N[i] = 1;
	}

	hipMalloc((void**) &d_M, size);
	hipMalloc((void**) &d_N, size);
	hipMalloc((void**) &d_P, size);

	hipMemcpy(d_M, h_M, size, hipMemcpyHostToDevice);
	hipMemcpy(d_N, h_N, size, hipMemcpyHostToDevice);

	dim3 dimBlock(16, 16, 1);
	dim3 dimGrid((width-1)/16+1, (width-1)/16+1, 1);

	MatrixMulKernel1<<<dimBlock, dimGrid>>>(d_M, d_N, d_P, width);

	hipMemcpy(h_P, d_P, size, hipMemcpyDeviceToHost);

	printf("\n");
	for (size_t i = 0; i < width*width; i++) {
		printf("%6f--", h_P[i]);
	}
	printf("\n");

	return 0;
}
