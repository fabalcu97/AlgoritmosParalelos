#include "hip/hip_runtime.h"

/*
compile using :

nvcc -std=c++11 -arch=sm_35 -DnumOfArrays=<number of arrays> -DmaxElements=<maximum number of elements per array> GPU-ArraySort.cu -o out


*/


/*
Copyright (C) Muaaz Gul Awan and Fahad Saeed
This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/



#include<iostream>
#include<vector>
#include<stdlib.h>
#include<time.h>
#include<math.h>
#include<ctime>
#include<algorithm>
#include<utility>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include<random>
using namespace std;

#define numOfArrays 50000
#define maxElements 4000
// #define numOfArrays 5
// #define maxElements 20
#define tempo 2
#define m 20
#define BUCKETS (maxElements/m)
#define sampleRate 10
#define SAMPLED (sampleRate*maxElements)/100
#define BLOCK_SIZE 1024

//data generation
template <typename mType>
struct dataArrays{
	vector<mType> dataList;
	int *prefixArray;
};


/* template <typename type>
dataArrays<type> dataGen (int numOfArrays, int maxArraySize, int minArraySize){

   dataArrays<int> data;
   data.prefixArray = new int[numOfArrays+1]; //exclusive prefix scan
   const int range_from = 0;
   const unsigned int range_to = 30;//2147483647; //2^31 - 1
   random_device rand_dev;
   mt19937 generator(rand_dev());
   uniform_int_distribution<int> distr(range_from, range_to);
   int prefixSum = 0;
   srand(time(0));
	for( int i = 0; i < numOfArrays; i++){

		int size = rand()%(maxArraySize-minArraySize + 1) + minArraySize;
		data.prefixArray[i] = prefixSum;
		for(int j = prefixSum; j < prefixSum + size; j++){
			data.dataList.push_back(distr(generator));
		}
		prefixSum += size;
	}

	data.prefixArray[numOfArrays] = prefixSum;
	return data;
} */


//swap function for Insertion sort
template <class type>
 __device__ void  swapD (type &a, type &b)

{
    /* &a and &b are reference variables */
    type temp;
        temp=a;
	a=b;
        b=temp;
}

//insertion sort
template <class type>
 __device__ void insertionSort(type *input, int begin, int end){
        int i, j; //,tmp;
        for (i = begin+1; i < end; i++) {
            j = i;
            while (j > begin && input[j - 1] > input[j]) {
                  swapD(input[j], input[j-1]);
                   j--;
                 }//end of while loop
           }
        }

__device__ int left(int index) {
		return (index << 1) + 1;
}
/*
template <typename type>
__device__ void maxSiftDown(type *array,int fromIndex, int toIndex, int index) {
	   int leftChildIndex = left(index);
	   // Right child index is one position from left child index towards
	   // larger indices.
	   int rightChildIndex = leftChildIndex + 1;
	   int maxChildIndex = index;
	   // Save the array component we want to sift down.
	   int target = array[fromIndex + index];

	   for (;;) {
		   if (fromIndex + leftChildIndex < toIndex
				   && array[fromIndex + leftChildIndex] > target) {
			   maxChildIndex = leftChildIndex;
		   }

		   if (maxChildIndex == index) {
			   if (fromIndex + rightChildIndex < toIndex
					   && array[fromIndex + rightChildIndex] > target) {
				   maxChildIndex = rightChildIndex;
			   }
		   } else {
			   if (fromIndex + rightChildIndex < toIndex
					   && array[fromIndex + rightChildIndex] >
						  array[fromIndex + leftChildIndex]) {
				   maxChildIndex = rightChildIndex;
			   }
		   }

		   if (maxChildIndex == index) {
			   // No swap. Just insert the sifted element.
			   array[fromIndex + maxChildIndex] = target;
			   return;
		   }

		   // No swap here neither.
		   // Just move up the maximum to current position.
		   array[fromIndex + index] = array[fromIndex + maxChildIndex];

		   index = maxChildIndex;
		   leftChildIndex = left(index);
		   rightChildIndex = leftChildIndex + 1;
	   }
}

template <class type>
 __device__ void buildMaxHeap(type *array, int fromIndex, int toIndex) {
    int rangeLength = toIndex - fromIndex;

    for (int i = rangeLength / 2; i >= 0; --i) {
 	   maxSiftDown(array, fromIndex, toIndex, i);
    }
 }

// main function to do heap sort
template <class type>
 __device__ void heapSort(type *array, int fromIndex, int toIndex) {
     if (toIndex - fromIndex < 2) {
         return;
     }

     // CLRS says 'BUILD-MAX-HEAP' is O(n).
     buildMaxHeap(array, fromIndex, toIndex);

     // And this is O(n log n).
     for (int i = toIndex - 1; i > fromIndex; --i) {
         int tmp = array[i];
         array[i] = array[fromIndex];
         array[fromIndex] = tmp;
         maxSiftDown(array, fromIndex, i, 0);
     }
 }*/

 template <class type>
  __device__ void heapify(type *arr, int n, int i, int ini)
 {
 	int largest = i;  // Initialize largest as root
 	i -= ini;
 	int l = 2*i + 1 + ini;  // left = 2*i + 1
 	int r = 2*i + 2 + ini;  // right = 2*i + 2
 	i += ini;

 	// If left child is larger than root
 	if (l < n && arr[l] > arr[largest]){
 		largest = l;
 	}
 	// If right child is larger than largest so far
 	if (r < n && arr[r] > arr[largest]){
 		largest = r;
 	}
 	// If largest is not root
 	if (largest != i){
 		swapD(arr[i], arr[largest]);

 		// Recursively heapify the affected sub-tree
 		heapify(arr, n, largest, ini);
 	}
 }

 // main function to do heap sort
 template <class type>
  __device__ void heapSort(type *arr, int ini, int fin, int n)
 {
 	// Build heap (rearrange array)
 	for (int i = ((fin+1-ini)/ 2) - 1+ini; i >= ini; i--){
 		heapify(arr, fin+1, i, ini);
 	}
 	//printArray(arr, ini, n);

 	// One by one extract an element from heap
 	for (int i=fin; i>=ini; i--){
 		// Move current root to en
 		swapD(arr[ini], arr[i]);

 		// call max heapify on the reduced heap
 		heapify(arr, i, ini, ini);
 		//printArray(arr, ini, n);
 	}
 }


 int findArr(float input[], int size, int key){
          for(int i = 0; i < size; i++)
            {
               if(input[i] == key)
                  return 2;

            }

return 0;

}

__device__ void getMinMax(float input[], int beginPtr, int endPtr, float *ret){
          float min = input[beginPtr];
          float max = 0;
        // int *ret = new int[2];
          for(int i = beginPtr; i < endPtr; i++){
              if(min > input[i])
                  min = input[i];
              if (max < input[i])
                  max = input[i];
            }

     ret[0] = min;
     ret[1] = max;
//return ret;

}

__device__ void getSplitters (float input[], float splitters[], int sample[], int beginPtr, int endPtr){
           __shared__ float mySamples[SAMPLED];
            float *ret = new float[2];
            for(int i = 0; i < SAMPLED; i++)
	   mySamples[i] = input[beginPtr+sample[i]];

	 insertionSort(mySamples, 0, SAMPLED);
       int splitterIndex = blockIdx.x*(BUCKETS+1)+1;
       int splittersSize=0;
	 for(int i = (SAMPLED)/(BUCKETS);splittersSize < BUCKETS-1; i +=SAMPLED/(BUCKETS)){
            splitters[splitterIndex] = mySamples[i];
            splitterIndex++;
            splittersSize++;
             }
          getMinMax(input, beginPtr, endPtr, ret);
           splitters[blockIdx.x*(BUCKETS+1)] = ret[0]-2;//to accodmodate the smallest
           splitters[blockIdx.x*(BUCKETS+1)+BUCKETS] = ret[1];

      delete [] ret;
}

__device__ void getBuckets2(float input[], float splitters[], int beginPtr, int endPtr, int bucketsSize[], float myInput[]){
      int id = threadIdx.x;
      int sizeOffset = blockIdx.x*BUCKETS+threadIdx.x;
      int bucketSizeOff = sizeOffset+1;
      float myBucket[maxElements];
      int indexSum=0;
      bucketsSize[bucketSizeOff] = 0;

     for(int i = 0; i < maxElements; i++){
         if(myInput[i] > splitters[id] && myInput[i] <= splitters[id+1]){
         myBucket[bucketsSize[bucketSizeOff]] = myInput[i];
         bucketsSize[bucketSizeOff]++;

}


     }

   __syncthreads();

         //prefix sum for bucket sizes of current array
         for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketsSize[blockIdx.x*BUCKETS+j+1];

         //writing back current buckt back to the input memory
	 for(int i = 0; i < bucketsSize[bucketSizeOff]; i++)
             input[indexSum+beginPtr+i] = myBucket[i];


}



__device__ void getBuckets(float input[], float splitters[], int beginPtr, int endPtr, int bucketsSize[]){
      int id = threadIdx.x;
      int sizeOffset = blockIdx.x*BUCKETS+threadIdx.x;
      int bucketSizeOff = sizeOffset+1;
      float myBucket[maxElements];
      int indexSum=0;
      bucketsSize[bucketSizeOff] = 0;

     for(int i = 0; i < maxElements; i++){
         if(input[beginPtr+i] > splitters[id] && input[beginPtr+i] <= splitters[id+1]){
         myBucket[bucketsSize[bucketSizeOff]] = input[beginPtr+i];
         bucketsSize[bucketSizeOff]++;

}


     }

   __syncthreads();

         //prefix sum for bucket sizes of current array
         for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketsSize[blockIdx.x*BUCKETS+j+1];

         //writing back current buckt back to the input memory
	 for(int i = 0; i < bucketsSize[bucketSizeOff]; i++)
             input[indexSum+beginPtr+i] = myBucket[i];


}
__device__ void bucketer(int input[], int bucketsSize[], int sample[], int beginPtr, int endPtr, int output[]){
          int id = blockIdx.x;
          const int toBeSampled = SAMPLED;
         const  int buckets = BUCKETS;

        __shared__ int splitters[buckets-1];


        //converting samples into unsorted-unselected-splitters

    for(int i = 0; i < toBeSampled; i ++)
        sample[i] = input[beginPtr+sample[i]];
           insertionSort(sample, 0, toBeSampled);

         //taking splitters out
          int splittersSize=0;
	 for(int i = (toBeSampled)/(buckets);splittersSize < buckets-1; i +=toBeSampled/(buckets)){
             splitters[splittersSize] = sample[i];
            splittersSize++;
             }


          int sumBsize=0;
          int sIndex = 0;
          for(int i = id*BUCKETS; i < (id*BUCKETS+BUCKETS); i++){
              bucketsSize[i] = 0;
              for(int j = 0; j <maxElements ; j++){
              //for bucket 0
               if(sIndex == 0){
                if( input[beginPtr+j] <= splitters[0]){
		   output[beginPtr+sumBsize+bucketsSize[i]]=input[beginPtr+j];
                   bucketsSize[i]++;
                   }
                    }
              //for last bucket
               else if(sIndex == buckets-1){
                    if( input[beginPtr+j] > splitters[splittersSize-1]){

		   output[beginPtr+sumBsize+bucketsSize[i]] = input[beginPtr+j];
                   bucketsSize[i]++;
                       }
                  }
               else{
                    if( input[beginPtr+j] > splitters[sIndex-1] && input[beginPtr+j] <= splitters[sIndex]) {
		   output[beginPtr+sumBsize+bucketsSize[i]] = input[beginPtr+j];
                   bucketsSize[i]++;
                }

                }

	         }
                sumBsize += bucketsSize[i];
                sIndex++;
	      }
}


__global__ void splitterKer(float *data, float *splitters, int *mySample){
          if(blockIdx.x < numOfArrays){
             int id = blockIdx.x;
             int arrBegin = id*maxElements;
	     int arrEnd = arrBegin + maxElements;

	     __shared__ int sampleSh[SAMPLED];

	     for(int i = 0; i < SAMPLED; i++)
	        sampleSh[i] = mySample[i];

	     getSplitters(data, splitters, sampleSh, arrBegin, arrEnd);

          }
		  data[0] = 9999;
     }


__global__ void bucketEM2(float *data, int *bucketSizes, float *splittersGlob){
    if(blockIdx.x < numOfArrays){
        bucketSizes[0] = 0;
        int bid = blockIdx.x;
        int tid = threadIdx.x;
         int leftOvers = maxElements%BUCKETS;
         int jmpFac = maxElements/BUCKETS;
        int gArrayStart = bid*maxElements+tid*jmpFac;
        int gArrayEnd = (tid==(BUCKETS-1))?(gArrayStart + jmpFac+leftOvers):(gArrayStart + jmpFac);
        int lArrayStart = tid*jmpFac;
        __shared__ float myInput [maxElements];

        int arrBegin = bid*maxElements;
        int arrEnd = arrBegin + maxElements;
        int splitterIndexSt = blockIdx.x*(BUCKETS+1);
        int splitterIndexEd = splitterIndexSt + BUCKETS+1;
        __shared__ float splitters[BUCKETS+1];
//copy my array in shared memory in parallel
           for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 myInput[i] = data[j];

        }
      __syncthreads();
        int j = 0;
        for(int i = splitterIndexSt; i < splitterIndexEd; i++){
           splitters[j] = splittersGlob[i];
           j++;
}

	getBuckets2(data, splitters, arrBegin, arrEnd, bucketSizes, myInput);

	}
}



__global__ void sortEM2(float *buckets, int *bucketSizes){
       if(blockIdx.x < numOfArrays && threadIdx.x < BUCKETS){
        int bid = blockIdx.x;
        int tid = threadIdx.x;
        int leftOvers = maxElements%BUCKETS;
        int jmpFac = maxElements/BUCKETS;
        int gArrayStart = bid*maxElements+tid*jmpFac;
         int gArrayEnd = (tid==(BUCKETS-1))?(gArrayStart + jmpFac+leftOvers):(gArrayStart + jmpFac);
        int lArrayStart = tid*jmpFac;

        __shared__ float myArray [maxElements];
        int indexSum = 0;


          for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 myArray[i] = buckets[j];

        }
        __syncthreads();
          for(int j = 0; j < threadIdx.x; j++)
            indexSum += bucketSizes[blockIdx.x*BUCKETS+j+1];


		//   insertionSort(myArray, indexSum,indexSum+ bucketSizes[blockIdx.x*BUCKETS+threadIdx.x+1]);
		//heapSort(myArray, indexSum, indexSum + bucketSizes[blockIdx.x*BUCKETS+threadIdx.x+1]);
		heapSort(myArray, indexSum, indexSum + bucketSizes[blockIdx.x*BUCKETS+threadIdx.x+1], maxElements);
          __syncthreads();


           for(int i=lArrayStart,j=gArrayStart;j<gArrayEnd;i++,j++){
                 buckets[j] = myArray[i];
        }
     __syncthreads();
}


}

__global__ void sortEM(int *buckets, int *prefixSum){

      if(blockIdx.x < numOfArrays && threadIdx.x < BUCKETS){
        int bid = blockIdx.x;
        int tid = threadIdx.x;

        int left =(tid)+bid*(BUCKETS);
        int right = (tid+1)+bid*(BUCKETS);

          insertionSort(buckets, prefixSum[left], prefixSum[right]);
	  }
}



int main ()
{

	const int range_from = 0;
	// const unsigned int range_to = 2147483647; //2^31 - 1
	const unsigned int range_to = 1024;
	random_device rand_dev;
	mt19937 generator(rand_dev());
	uniform_int_distribution<int> distr(range_from, range_to);
	size_t f,t;
	int *d_bucketSizes , *h_bucketSizes;

	float *d_data, *h_buckets, *d_splitters, *h_splitters;
	int numBlocks = ceil((float)(BUCKETS*numOfArrays+1)/(BLOCK_SIZE<<1));
	dim3 dimGrid(numBlocks, 1, 1);
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	float *h_data = new float[numOfArrays*maxElements];
	h_buckets = new float[numOfArrays*maxElements];
	h_bucketSizes = new int[BUCKETS*numOfArrays+1];
	h_splitters = new float[(BUCKETS+1)*sizeof(float)*numOfArrays];
	size_t size_heap, size_stack;
	int *h_sample = new int[SAMPLED];
	int *d_sample;
	//hipSetDevice(0);
	hipMemGetInfo(&f, &t);

	//setting stack size limit
	hipDeviceSetLimit(hipLimitStackSize,10240);
	hipDeviceGetLimit(&size_heap, hipLimitMallocHeapSize);
	hipDeviceGetLimit(&size_stack, hipLimitStackSize);

	//generating regular samples
	int max = maxElements;
	int  sam = SAMPLED;
	int stride = max/sam;
	int sampleVal = 0;
	for( int i = 0; i < SAMPLED; i++){
		h_sample[i] = sampleVal;
		sampleVal += stride;
	}


	// allocating device memory for data, sampled indices and bucket sizes
	hipMalloc((void**) &d_sample, SAMPLED*sizeof(float));
	hipMalloc((void**) &d_data, numOfArrays*maxElements*sizeof(float));
	hipMalloc((void**) &d_bucketSizes, numOfArrays*sizeof(int)*BUCKETS+sizeof(int));
	hipMalloc((void**) &d_splitters, (BUCKETS+1)*sizeof(float)*numOfArrays);
	srand(time(NULL));
	hipMemGetInfo(&f,&t);

	//new data gens
	//cout<<"OJO"<<endl;
	for(int i = 0; i < numOfArrays; i++){
		for(int j = 0; j < maxElements; j++){
			h_data [j+i*maxElements] = distr(generator) ;
			//cout<<h_data [j+i*maxElements]<<",";
		}
		//cout<<endl;
	}

	//copy data and samples to GPU
	hipMemcpy(d_data, h_data, numOfArrays*maxElements*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_sample, h_sample, SAMPLED*sizeof(float), hipMemcpyHostToDevice);

	clock_t firstKrTime = clock();
	splitterKer<<<numOfArrays,1>>>(d_data, d_splitters, d_sample);
	hipDeviceSynchronize();
	firstKrTime = clock() - firstKrTime;

	hipError_t errSync  = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess){
		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));
	}
	if (errAsync != hipSuccess){
		printf("Async kernel error: %s\n", hipGetErrorString(errAsync));
	}

	hipMemcpy(h_splitters, d_splitters, (BUCKETS+1)*sizeof(float)*numOfArrays, hipMemcpyDeviceToHost);

	clock_t secondKrTime = clock();
	//cout<<secondKrTime<<endl;
	bucketEM2<<<numOfArrays,BUCKETS>>>(d_data, d_bucketSizes, d_splitters);
	hipDeviceSynchronize();
	secondKrTime = clock()-secondKrTime;
	//cout<<secondKrTime<<endl;
	//cout<<"--------"<<endl;
	hipMemGetInfo(&f,&t);

	//copying bucket sizes from first kernel back to cpu for prefix sum, to be replaced with prefix sum code
	hipMemcpy(h_bucketSizes, d_bucketSizes, sizeof(int)*(BUCKETS*numOfArrays+1), hipMemcpyDeviceToHost);
	//freeing the sample indices memory space and bucket sizes memory
	hipFree(d_sample);
	hipFree(d_splitters);

	clock_t fourKrTime = clock();
	sortEM2<<<numOfArrays, BUCKETS>>>(d_data, d_bucketSizes);

	hipDeviceSynchronize();

	fourKrTime = clock()-fourKrTime;

	cout<<(firstKrTime+secondKrTime+fourKrTime)/double(CLOCKS_PER_SEC)*1000<<endl;
	//copying the sorted data back
	hipMemcpy(h_buckets, d_data, numOfArrays*maxElements*sizeof(float), hipMemcpyDeviceToHost);

	// for(int i = 0; i < numOfArrays; i++){
	// 	for(int j = 0; j < maxElements; j++){
	// 		cout<<h_buckets[j+i*maxElements]<<"-";
	// 	}
	// 	cout<<endl;
	// }

	//freeing the space for prefixSum and sorted data
	hipFree(d_data);
	hipFree(d_bucketSizes);

	free(h_bucketSizes);
	free(h_buckets);
	free(h_data);
	free(h_sample);
	free(h_splitters);
	return 0;
}
