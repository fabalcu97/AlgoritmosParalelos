#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>

using namespace std;
using namespace cv;

extern "C" void gray_parallel(unsigned char* h_in, unsigned char* h_out, int elems, int rows, int cols);

int main(){

	Mat d_image;

	//checkCudaErrors(hipFree(0));

	Mat image;
	image = imread("via.png",CV_LOAD_IMAGE_COLOR);

	namedWindow( "Display window", WINDOW_AUTOSIZE );
	imshow( "Display window", image );

	const int rows = image.rows;
	const int cols = image.cols;
	int elems = rows*cols*3;
	unsigned char *h_in = image.data;
	unsigned char *h_out = new unsigned char[rows*cols];

	gray_parallel(h_in, h_out, elems, rows, cols);

	Mat gray2 = Mat(rows,cols,CV_8UC1,h_out);

	namedWindow( "Display window GrayScale", WINDOW_AUTOSIZE );
	imshow( "Display window GrayScale", gray2 );
	waitKey(0);

	return 0;
}

__global__ void kernel(unsigned char* d_in, unsigned char* d_out, int w, int h){

    //int idx = blockIdx.x;
	//int idy = threadIdx.x;

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	//int gray_adr = idx*w + idy;
	int gray_adr =row*w+col;
	int clr_adr = 3*gray_adr;

	if(gray_adr<(w*h))
		{
			double gray_val = 0.21f*d_in[clr_adr] + 0.71f*d_in[clr_adr+1] + 0.07f*d_in[clr_adr+2];
			d_out[gray_adr] = (unsigned char)gray_val;
			//printf(" %d:%d=[%d,%d,%d,%d] \n", idx,idy,d_in[clr_adr],d_in[clr_adr+1],d_in[clr_adr+2],(int)gray_val);
		}
}

//   Kernel Calling Function

extern "C" void gray_parallel(unsigned char* h_in, unsigned char* h_out, int elems, int rows, int cols){

	unsigned char* d_in;
	unsigned char* d_out;
	hipMalloc((void**) &d_in, elems);
	hipMalloc((void**) &d_out, rows*cols);

	dim3 dimBlock(96, 96, 1);
	dim3 dimGrid((cols-1)/96+1, (rows-1)/96+1, 1);

	hipMemcpy(d_in, h_in, elems*sizeof(unsigned char), hipMemcpyHostToDevice);
    //kernel<<<rows,cols>>>(d_in, d_out, cols, rows);
	kernel<<<dimBlock, dimGrid>>>(d_in, d_out, cols, rows);

	hipError_t errSync  = hipGetLastError();
	if (errSync != hipSuccess)
  		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	hipMemcpy(h_out, d_out, rows*cols*sizeof(unsigned char), hipMemcpyDeviceToHost);
	hipFree(d_in);
	hipFree(d_out);
}
