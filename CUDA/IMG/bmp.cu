#include "hip/hip_runtime.h"
#include "bitmap_image.hpp"

__global__
void GScale(bitmap_image* img, int iRow, int iCol){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if (col < iCol && row < iRow){
		img.set_pixel(col, row, 0.07, 0.71, 0.21);
	}
}

int main(int argc, char const *argv[]) {

	bitmap_image image("input.bmp");

	if (!image)
	{
		printf("Error - Failed to open: input.bmp\n");
		return 1;
	}

	image.save_image("output.bmp");
	return 0;
}
