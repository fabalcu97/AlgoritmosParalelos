#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>

using namespace std;
using namespace cv;

__global__
void GScale(Vec3b* img, int iRow, int iCol){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if (col < iCol && row < iRow){

		img[row][col][0] *= 0.07f;		//BLUE
		img[row][col][1] *= 0.71f;		//GREEN
		img[row][col][2] *= 0.21f;		//RED
	}
}

int main(){

	Mat d_image;

	Mat image;
	image = imread("tux.bmp", CV_LOAD_IMAGE_COLOR);

	namedWindow( "Display window", WINDOW_AUTOSIZE );
	imshow( "Display window", image );

	Vec3b aa;
	Vec3b d_aa;

	for(int y = 0; y < image.rows; y++) {
		Vec3b tmp;
		for(int x = 0; x < image.cols; x++) {
			tmp.push_back(image.at<Vec3b>(y, x));
		}
		aa.push_back(tmp);
	}

	hipMalloc( (void**) &d_aa, sizeof(Mat));
	hipMemcpy(&d_aa, &aa, sizeof(uchar * image.rows * image.cols), hipMemcpyHostToDevice);

	/* for(int y = 0; y < image.rows; y++) {
		for(int x = 0; x < image.cols; x++) {
			image.at<Vec3b>(y, x)[0] *= 0.3;		//BLUE
			image.at<Vec3b>(y, x)[1] *= 0.3;		//GREEN
			image.at<Vec3b>(y, x)[2] *= 0.3;		//RED
		}
	}
	*/

	GScale<<<1, 96>>>(&aa, image.rows, image.cols);
	hipMemcpy(&aa, &d_aa, sizeof(uchar * image.rows * image.cols), hipMemcpyDeviceToHost);


	namedWindow( "Display window GrayScale", WINDOW_AUTOSIZE );
	imshow( "Display window GrayScale", image );
	waitKey(0);

	return 0;
}
