#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>

using namespace std;
using namespace cv;

__global__
void GScale(unsigned char*  d_a, unsigned char* d_b, int iRow, int iCol){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;

	if (col < iCol && row < iRow){
		int clr_adr=(row*iCol+col)*3;
		/*img[row][col][0] *= 0.07f;		//BLUE
		img[row][col][1] *= 0.71f;		//GREEN
		img[row][col][2] *= 0.21f;		//RED*/
		double gray_val = 0.21f*d_a[clr_adr] + 0.71f*d_a[clr_adr+1] + 0.07f*d_a[clr_adr+2];
		d_b[(row*iCol+col)] = (unsigned char)gray_val;
		d_b[0]=(unsigned char)100.0;
	}
}

int main(){

	Mat d_image;

	//checkCudaErrors(hipFree(0));

	Mat image;
	image = imread("input.bmp",CV_LOAD_IMAGE_COLOR);

	namedWindow( "Display window", WINDOW_AUTOSIZE );
	imshow( "Display window", image );

	unsigned char *a,*b,*d_a,*d_b;

	a = (unsigned char*)malloc(60*50*sizeof(unsigned char));
	b = (unsigned char*)malloc(60*50*sizeof(unsigned char));

	hipMalloc((void**)&d_a, 60*50*3);
    hipMalloc((void**)&d_b, 60*50);

	a=image.data;

	hipMemcpy(&d_a, &a, sizeof(unsigned char)* 3 * 60 * 50, hipMemcpyHostToDevice);

	/* for(int y = 0; y < image.rows; y++) {
		for(int x = 0; x < image.cols; x++) {
			image.at<Vec3b>(y, x)[0] *= 0.3;		//BLUE
			image.at<Vec3b>(y, x)[1] *= 0.3;		//GREEN
			image.at<Vec3b>(y, x)[2] *= 0.3;		//RED
		}
	}
	*/

	for(int i = 0; i < image.rows*image.cols; i++)
		printf("%d  - ",a[i]);

	dim3 dimBlock(26,26,1);
	dim3 dimGrid((50-1)/26+1, (60-1)/26+1, 1);

	GScale<<<60, 50>>>(d_a, d_b, 60, 50);

	hipError_t errSync  = hipGetLastError();
	if (errSync != hipSuccess)
  		printf("Sync kernel error: %s\n", hipGetErrorString(errSync));

	hipMemcpy(&b, &d_b, sizeof(uchar) * 60 * 50, hipMemcpyDeviceToHost);

	printf("%d\n",b[0]);

	for(int i = 0; i < image.rows*image.cols; i++)
		printf("%d  - ",b[i]);

	Mat gray = Mat(image.rows, image.cols, CV_8UC1, b);

	hipFree(d_a);
	hipFree(d_b);

	namedWindow( "Display window GrayScale", WINDOW_AUTOSIZE );
	imshow( "Display window GrayScale", gray );
	waitKey(0);

	return 0;
}
