
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__
void saxpy(int n, float *x, float *y, float *c)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;

	if (i < n){
		c[i] = x[i] + y[i];
	}
}

__host__
int main(void)
{

	int N = 300;
	float *x, *y, *c, *d_x, *d_y, *d_c;

	x = (float*)malloc(N*sizeof(float));
	y = (float*)malloc(N*sizeof(float));
	c = (float*)malloc(N*sizeof(float));

	hipMalloc( (void**) &d_x, N*sizeof(float));
	hipMalloc( (void**) &d_y, N*sizeof(float));
	hipMalloc( (void**) &d_c, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}

	hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

	saxpy<<<1, N>>>(N, d_x, d_y, d_c);

	hipMemcpy(c, d_c, N*sizeof(float), hipMemcpyDeviceToHost);

	// float maxError = 0.0f;

	for (int i = 0; i < N; i++){
		printf("%f - ", c[i]);
	}

	//   maxError = max(maxError, abs(y[i]-4.0f));
	// printf("Max error: %f\n", maxError);
}
