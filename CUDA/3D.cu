#include "hip/hip_runtime.h"
#include <stdio.h>

using namespace std;

#define BLOCK_SIZE 16
#define GRID_SIZE 1

__global__
void GScale(float* img, int iRow, int iCol, int id){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int a = blockIdx.z*blockDim.z + threadIdx.z;

	if (col < iCol && row < iRow && a < id){

		img[row*iCol+col+a*iRow*iCol]=2*img[row*iCol+col+a*iRow*iCol];
	}
}

__host__
int main(void)
{
  int N = 60;
  float *x, *d_x;

  x = (float*)malloc(N*N*N*sizeof(float));

  hipMalloc(&d_x, N*N*N*sizeof(float));

  for (int i = 0; i < N*N*N; i++) {
    x[i] = 1.0f;
  }

	dim3 dimBlock(16,16,16);
	dim3 dimGrid((N-1)/16+1, (N-1)/16+1, (N-1)/16+1);

  hipMemcpy(d_x, x, N*N*N*sizeof(float), hipMemcpyHostToDevice);

  GScale<<<dimGrid, dimBlock>>>(&d_x, N, N, N);

  hipMemcpy(x, d_x, N*N*N*sizeof(float), hipMemcpyDeviceToHost);

  	for (int i = 0; i < N*N*N; i++){
		printf("%f - ", x[i]);
	}

}
