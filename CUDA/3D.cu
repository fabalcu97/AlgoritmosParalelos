
#include <hip/hip_runtime.h>
#include <stdio.h>

using namespace std;

#define BLOCK_SIZE 16
#define GRID_SIZE 1

__global__
void GScale(float* img, float* res, int iRow, int iCol, int id){

	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int a = blockIdx.z*blockDim.z + threadIdx.z;

	if (col < iCol && row < iRow && a<id){

		res[row*iCol+col+a*iCol*iRow]=2.0*img[row*iCol+col+a*iCol*iRow];
	}
}

__host__
int main(void)
{
  int N = 6;
  float *x, *y, *d_x, *d_y;

  x = (float*)malloc(N*N*N*sizeof(float));
  y = (float*)malloc(N*N*N*sizeof(float));

  hipMalloc((void**)&d_x, N*N*N*sizeof(float));
  hipMalloc((void**)&d_y, N*N*N*sizeof(float));

  for (int i = 0; i < N*N*N; i++) {
    x[i] = 5.0;
	//y[i] = 0.0;
  }

	dim3 dimBlock(16,16,16);
	dim3 dimGrid((N-1)/16+1, (N-1)/16+1, (N-1)/16+1);
	//dim3 dimGrid(1, 1, 1);


  hipMemcpy(d_x, x, N*N*N*sizeof(float), hipMemcpyHostToDevice);

  GScale<<<dimBlock, dimGrid>>>(d_x, d_y, N, N, N);

  hipMemcpy(y, d_y, N*N*N*sizeof(float), hipMemcpyDeviceToHost);

  	for (int i = 0; i < N*N*N; i++){
		printf("%f - ", y[i]);
	}

	hipFree(d_x);
	hipFree(d_y);

}
